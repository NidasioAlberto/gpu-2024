#include <hip/hip_runtime.h>
#define N (1 << 10)

__global__ void vector_add(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int h_va[N], h_vb[N], h_vc[N];
    int *d_va, *d_vb, *d_vc;

    // Data initialization
    for (int i = 0; i < N; i++)
    {
        h_va[i] = i % 6;
        h_vb[i] = i % 8;
    }

    // Device memory allocation
    hipMalloc(&d_va, N * sizeof(int));
    hipMalloc(&d_vb, N * sizeof(int));
    hipMalloc(&d_vc, N * sizeof(int));

    // CPU -> GPU data transmission
    hipMemcpy(d_va, h_va, N * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_vb, h_vb, N * sizeof(int),
               hipMemcpyHostToDevice);

    // Kernel launch
    dim3 blocksPerGrid(N / 256, 1, 1);
    dim3 threadsPerBlock(256, 1, 1);
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_va, d_vb,
                                                   d_vc);

    // GPU->CPU data transmission
    hipMemcpy(h_vc, d_vc, N * sizeof(int),
               hipMemcpyDeviceToHost);

    // Device memory freeing
    hipFree(d_va);
    hipFree(d_vb);
    hipFree(d_vc);

    return 0;
}
